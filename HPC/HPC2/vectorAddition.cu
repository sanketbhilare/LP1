
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    
    c[id] = a[id] + b[id];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%size;
	}
}



int main()
{
    int N=100000;
    
    int size = N * sizeof(int);
    
    int *A, *B, *C;
    
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);
    
    int *Ad, *Bd, *Cd;
    
    random_ints(A,N);
    random_ints(B,N);
    
    hipMalloc(&Ad, size);
    hipMalloc(&Bd, size);
    hipMalloc(&Cd, size);
    
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    
    dim3 blockSize(1024,1);
    dim3 gridSize(1,1);
    
    add <<<gridSize , blockSize>>>(Ad, Bd, Cd, N);
    
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    
    for(int i=0;i<5;i++)
    {
        printf("%d\t%d\t%d\n",A[i], B[i], C[i]);
    }
    
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
    
    
}