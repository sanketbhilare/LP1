
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define size 10

__global__ void vectMatMul(int *a, int *b, int*c, int n)
{
    int id = threadIdx.x;
    if(id<n)
    {
      for( int j=0;j<size; j++)
      {
          c[id] = c[id] + (a[j] * b[id*n + j]);
      }
    }
}




int main()
{
    int *A,*B,*C;
    A = (int*)malloc(size * sizeof(int));
    B = (int*)malloc(size * size * sizeof(int));
    C = (int*)malloc(size * sizeof(int));
    
    
    for(int i=0; i<size;i++)
    {
        A[i] = rand()%10;
        for(int j=0; j<size; j++)
        {
            *(B + i*size + j) = rand()%10;
        }
    }
    
    
    int *AD, *BD, *CD;
    
    hipMalloc(&AD, size*sizeof(int));
    hipMalloc(&BD, size*size*sizeof(int));
    hipMalloc(&CD, size*sizeof(int));
    
    hipMemcpy(AD, A, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(BD, B, size*size*sizeof(int), hipMemcpyHostToDevice);
    
    vectMatMul<<<1,size>>>(AD, BD, CD, size);
    
    hipMemcpy(C, CD, size*sizeof(int), hipMemcpyDeviceToHost);
    
    
    
    printf("Vector: \n");
	for (int i = 0; i < size; i++)
	{
		printf("%d ", A[i]);
	}
	printf("\n");
	printf("Matrix: \n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%d ", *(B + i*size + j));
		}
		printf("\n");
	}
	printf("Product: \n");
	for (int i = 0; i < size; i++)
	{
		printf("%d ", C[i]);
	}
	printf("\n");
    
    
    hipFree(AD);
    hipFree(BD);
    hipFree(CD);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
