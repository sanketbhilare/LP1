
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
#include<stdlib.h>
#include<time.h>

#define N 2048
using namespace std;

void random_ints(int *vector, int size){
    for(int i=0; i<size; i++)
        vector[i] = rand()%10;
}

void copy_int_to_float(float *dest, int *src, int size){
    for(int i=0; i<size; i++)
        dest[i] = float(src[i]);
}

__global__ void min(int *vector){
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0){
        if(tid < number_of_threads){
            int first_index = tid * step_size *2;
            int second_index = first_index + step_size;
            vector[first_index] = vector[first_index] > vector[second_index] ? vector[second_index] : vector[first_index];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void max(int *vector){
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0){
        if(tid < number_of_threads){
            int first_index = tid * step_size *2;
            int second_index = first_index + step_size;
            vector[first_index] = vector[first_index] < vector[second_index] ? vector[second_index] : vector[first_index];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void sum(int *vector){
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0){
        if(tid < number_of_threads){ //If thread is alive
            int first_index = tid * step_size * 2; //As each thread operates on 2 elements.
            int second_index = first_index + step_size;
            
            vector[first_index] += vector[second_index];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void sum_floats(float *vector){
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0){
        if(tid < number_of_threads){ //If thread is alive
            int first_index = tid * step_size * 2; //As each thread operates on 2 elements.
            int second_index = first_index + step_size;
            
            vector[first_index] += vector[second_index];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

__global__ void mean_diff_sq(float *vector, float mean){ //Calculates (x - x')^2
    vector[threadIdx.x] -= mean;
    vector[threadIdx.x] *= vector[threadIdx.x];
}

int main(void){
    int size = N * sizeof(int);
    int *vec; //Host copy of vec
    int *d_vec; //Device copy of vec
    int result;
    
    srand(time(0));

    vec = (int *)malloc(size);
    random_ints(vec, N);

    hipMalloc((void **)&d_vec, size);
    
    //SUM
    hipMemcpy(d_vec, vec, size, hipMemcpyHostToDevice);
    sum<<<1, N/2>>>(d_vec);
    //Copy the first element of array back to result
    hipMemcpy(&result, d_vec, sizeof(int), hipMemcpyDeviceToHost);
    printf("Sum is: %d", result);


    //MIN
    hipMemcpy(d_vec, vec, size, hipMemcpyHostToDevice);
    min<<<1, N/2>>>(d_vec);
    //Copy the first element of array back to result
    hipMemcpy(&result, d_vec, sizeof(int), hipMemcpyDeviceToHost);
    printf("\\nMin is: %d", result);
    
    
    //MAX
    hipMemcpy(d_vec, vec, size, hipMemcpyHostToDevice);
    max<<<1, N/2>>>(d_vec);
    //Copy the first element of array back to result
    hipMemcpy(&result, d_vec, sizeof(int), hipMemcpyDeviceToHost);
    printf("\\nMax is: %d", result);
    
    
    //MEAN
    hipMemcpy(d_vec, vec, size, hipMemcpyHostToDevice);
    sum<<<1, N/2>>>(d_vec);
    //Copy the first element of array back to result
    hipMemcpy(&result, d_vec, sizeof(int), hipMemcpyDeviceToHost);
    float mean = float(result)/N;
    printf("\\nMean is: %f", mean);
    
    
    //STD. DEV
    float *float_vec;
    float *d_float_vec;
    
    float_vec = (float *)malloc(N*sizeof(float));
    hipMalloc((void **)&d_float_vec, N*sizeof(float));
    
    copy_int_to_float(float_vec, vec, N);
    
    hipMemcpy(d_float_vec, float_vec, N*sizeof(float), hipMemcpyHostToDevice);
    
    mean_diff_sq<<<1, N>>>(d_float_vec, mean);
    sum_floats<<<1, N/2>>>(d_float_vec);
    
    float res;
    hipMemcpy(&res, d_float_vec, sizeof(res), hipMemcpyDeviceToHost);
    
    res /= N;
    printf("\\nVariance: %f", res);
    res = sqrt(res);
    printf("\\nStd. Dev: %f", res);
    
    
    //Free allocated memory
    hipFree(d_vec);
    
    printf("\\n");
    return 0;
}